#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ctime>

#define BLOCK_SIZE 64

double* matrix_transpose(double* matrix, int n);
void mat_mul(double* a, double* b, double* c, int n);


void restore_q(double* q, double* restored_q, int n) {
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            restored_q[n * i + j] = (double)(i == j);
     for (int jb = n - 1; jb >= 0; jb -= BLOCK_SIZE) {
        for (int ib = n - 1; ib >= 0; ib -= BLOCK_SIZE) {
            for (int j = 0; j < BLOCK_SIZE; ++j) {
                for (int i = 0; i < BLOCK_SIZE; ++i) {
                    int row_abs = ib - i;
                    int col_abs = jb - j;
                    if (row_abs > col_abs) {
                        double c = q[row_abs*n + col_abs];
                        double s = -q[col_abs*n + row_abs];
                        for (int k = 0; k < n; ++k) {
                            double q0k =  restored_q[col_abs*n + k] * c + restored_q[row_abs*n + k] * s;
                            double q1k = -restored_q[col_abs*n + k] * s + restored_q[row_abs*n + k] * c;
                            restored_q[col_abs*n + k] = q0k;
                            restored_q[row_abs*n + k] = q1k;
                        }
                    }
                }
            }
        }
    }
}

bool check_result(double* A, double *q, double *R, int size) {
    const double eps = 1e-10;
    double *Q = new double[size*size];
    restore_q(q, Q, size);
    bool OK = true;

    for (int i = 0; i < size && OK; ++i) {
        for (int j = 0; j < i && OK; ++j) {
            if (R[i*size + j] > eps) {
                // printf("R is not upper triangular\n");
                OK = false;
            }
        }
    }

    double *Qt = new double[size*size];
    memcpy(Qt, Q, sizeof(double)*size*size);
    matrix_transpose(Qt, size);

    double *QtQ = new double[size*size];
    mat_mul(Qt, Q, QtQ, size);

    for (int i = 0; i < size && OK; ++i) {
        for (int j = 0; j < i && OK; ++j) {
            if (std::abs((QtQ[i*size + j] - (i==j))) > eps) {
                // printf("Q^T*Q != I\n")
                OK = false;
            }
        }
    }

    double *QR = new double[size*size];
    mat_mul(Q, R, QR, size);

    for (int i = 0; i < size && OK; ++i) {
        for (int j = 0; j < i && OK; ++j) {
            if (std::abs(QR[i*size + j] - A[i*size + j]) > eps) {
                // printf("Q^T*Q != I\n")
                OK = false;
            }
        }
    }

    delete[] Q;
    delete[] Qt; 
    delete[] QtQ;
    delete[] QR;

    return !OK;
}

void fillMatrix(double* matrix, int n) {
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            matrix[n * i + j] = (double)(rand() % 1000) / 100;
}

double* matrix_transpose(double* matrix, int n) {
    double d;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < i; ++j) {
            d = matrix[n * i + j];
            matrix[n * i + j] = matrix[n * j + i];
            matrix[n * j + i] = d;
        }
    }
    return matrix;
}

void mat_mul(double* a, double* b, double* c, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            c[n * i + j] = 0;
            for (int k = 0; k < n; ++k)
                c[n * i + j] += a[n * i + k] * b[n * k + j];
        }
    }
}


int sign(double x) {
    return (x > 0) - (x < 0);
}

bool isUpperTriangular(double *a, int size) {
    double eps = 1e-10;
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < i; ++j) {
            if (a[i*size + j] > eps)
                return false;
        }
    }
    return true;
}

void pm(double *a, int m=8, int n=8) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%8.4f ", a[i*n + j]);
        }
        std::cout << "\n";
    }
    std::cout << "-------------------\n";
}

__global__
void to_d_buf(double *d_a, double *d_buf, int i0, int j0, int size_d_a) {
    int i = i0 + BLOCK_SIZE*blockIdx.y  + threadIdx.y;
    int j = j0 + threadIdx.x;
    d_buf[(i-i0)*BLOCK_SIZE + (j-j0)] = d_a[i*size_d_a + j];
}


__global__
void upd(double *d_a, int size_a, double *d_cos, double *d_sin, int j) {
    int my_col = (j+BLOCK_SIZE) + blockIdx.x*BLOCK_SIZE + threadIdx.x;
    for (int i = j; i < size_a; i += BLOCK_SIZE) {
        for (int jj = 0; jj < BLOCK_SIZE; ++jj) {
            for (int ii = 0; ii < BLOCK_SIZE; ++ii) {
                int col = j + jj;
                int row = i + ii;
                if (row > col) {
                    double c =  d_cos[(row-j)*BLOCK_SIZE + jj];
                    double s = -d_sin[(row-j)*BLOCK_SIZE + jj];
                    // std::cout << c << ' ' << s << ' ' << row << ' ' << col << "\n";
                    double a0k =  d_a[col*size_a + my_col] * c + d_a[row*size_a + my_col] * s;
                    double a1k = -d_a[col*size_a + my_col] * s + d_a[row*size_a + my_col] * c;
                    d_a[col*size_a + my_col] = a0k;
                    d_a[row*size_a + my_col] = a1k;
                }
            }
        }
    }
}

void qr_cuda(double *a, double *q, int size) {
    double *buf = new double[size*BLOCK_SIZE * 3]; // a, q_cos, q_sin
    double *d_a;
    double *d_buf;
    double *d_cos;
    double *d_sin;
    hipMalloc(&d_a, sizeof(double)*size*size);
    hipMalloc(&d_cos, sizeof(double)*size*BLOCK_SIZE);
    hipMalloc(&d_sin, sizeof(double)*size*BLOCK_SIZE);
    hipMalloc(&d_buf, sizeof(double)*size*BLOCK_SIZE);
    hipMemcpy(d_a, a, sizeof(double)*size*size, hipMemcpyHostToDevice);

    hipEvent_t syncEvent;
    hipEventCreate(&syncEvent);

    for (int j = 0; j < size; j += BLOCK_SIZE) {
        for (int i = j; i < size; i += BLOCK_SIZE) {
            for (int jj = 0; jj < BLOCK_SIZE; ++jj) {
                int col = j + jj;
                for (int ii = 0; ii < BLOCK_SIZE; ++ii) {
                    int row = i + ii;
                    if (row > col) {
                        double a0 = a[col*size + col];
                        double a1 = a[row*size + col];
                        double sigma = sign(a1);
                        if (std::abs(a0) > std::abs(a1))
                            sigma = sign(a0);
                        double den = sigma * std::sqrt(a0*a0 + a1*a1);
                        double c = a0 / den;
                        double s = a1 / den;
                        // q[row*size + col] =  c;
                        // q[col*size + row] = -s;
                        int c_ind = (size*BLOCK_SIZE) + (row-j)*BLOCK_SIZE + (col-j);
                        int s_ind = c_ind + size*BLOCK_SIZE;
                        buf[c_ind] =  c;
                        buf[s_ind] = -s;
                        for (int k = col; k < j+BLOCK_SIZE; ++k) {
                            double a0k =  a[col*size + k] * c + a[row*size + k] * s;
                            double a1k = -a[col*size + k] * s + a[row*size + k] * c;
                            a[col*size + k] = a0k;
                            a[row*size + k] = a1k;
                        }
                    }
                }
            }
        }
        // cos sin to device
        hipMemcpy(d_cos, &buf[size*BLOCK_SIZE], sizeof(double)*size*BLOCK_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(d_sin, &buf[2*size*BLOCK_SIZE], sizeof(double)*size*BLOCK_SIZE, hipMemcpyHostToDevice);

        dim3 gridSize = dim3((size-j-BLOCK_SIZE)/BLOCK_SIZE, 1, 1);
        dim3 blockSize = dim3(BLOCK_SIZE, 1, 1);
        hipEventRecord(syncEvent, 0);
        upd<<<gridSize, blockSize>>>(d_a, size, d_cos, d_sin, j);

        for (int i = j; i < size; i += BLOCK_SIZE) { // cos&sin из buf в q
            for (int jj = 0; jj < BLOCK_SIZE; ++jj) {
                int col = j + jj;
                for (int ii = 0; ii < BLOCK_SIZE; ++ii) {
                    int row = i + ii;
                    if (row > col) {
                        int c_ind = (size*BLOCK_SIZE) + (row-j)*BLOCK_SIZE + (col-j);
                        int s_ind = c_ind + size*BLOCK_SIZE;
                        double c = buf[c_ind];
                        double s = buf[s_ind];
                        q[row*size + col] = c;
                        q[col*size + row] = s;
                    }
                }
            }
        }

        hipEventSynchronize(syncEvent);

        // один столбцовый блок из device в host
        gridSize = dim3(1, size/BLOCK_SIZE, 1);
        blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
        hipEventRecord(syncEvent, 0);
        to_d_buf<<<gridSize, blockSize>>>(d_a, d_buf, 0, j+BLOCK_SIZE, size);
        hipEventSynchronize(syncEvent);
        hipMemcpy(buf, d_buf, sizeof(double)*size*BLOCK_SIZE, hipMemcpyDeviceToHost);
        for (int ii = 0; ii < size; ++ii) {
            for (int jj = j+BLOCK_SIZE; jj < j + 2*BLOCK_SIZE; ++jj) {
                a[ii*size + jj] = buf[ii*BLOCK_SIZE + (jj-j-BLOCK_SIZE)];
            }
        }
    }
    delete[] buf;
    hipFree(d_a);
    hipFree(d_cos);
    hipFree(d_sin);
    hipFree(d_buf);
}

int main() {
    printf("%-5s\t%-10s\t%12s\n", "size", "time (s.)", "check_result");
    printf("---------------------------------\n");
    for (int size = 256; size <= 2048; size *= 2) {
        double *a = new double[size*size];
        double *q = new double[size*size];
        fillMatrix(a, size);
        double *r = new double[size*size];
        memcpy(r, a, sizeof(double)*size*size);

        qr_cuda(r, q, size);

        printf("%-5s\t%-10s\t", size, "time (s.)");
        if (!check_result(a, q, r, size)) {
            printf("%12s", "OK\n");
        }
        else {
            printf("%12s", "Error!\n");
        }
        

    }
    
    std::cout << isUpperTriangular(a, size) << "\n";


    return 0;
}
